#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange_fourth_order.go for more details.

///////////////////////////////////////////////////
// Important note: Currently only works for Nz=1 //
///////////////////////////////////////////////////

extern "C" __global__ void
addexchangefourthorder(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ I1, float* __restrict__ I2,
            uint8_t* __restrict__ regions,
            float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int    I  = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3  B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float I1__;  // second-order exchange stiffness
    float I2__;  // fourth-order exchange stiffness


    //////////////////
    // Central Spin //
    //////////////////
    i_    = idx(ix, iy, iz);
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                   // load m
    m_    = ( is0(m_)? m0: m_ );                                   // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    += 4 * I1__ * (1/(cx*cx) + 1/(cy*cy)) * m_;
    B    -= 12 * I2__ * (1/(cx*cx*cx*cx) + 1/(cy*cy*cy*cy)) * m_;
    B    -= 16 * I2__ / (cx*cx*cy*cy) * m_;


    ///////////////////////////////
    // Direct Nearest Neighbours //
    ///////////////////////////////
    
    // Left neighbour
    i_    = idx(lclampx(ix-1), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I1__ / (cx*cx)) * m_;
    B    += (8 * I2__ / (cx*cx*cx*cx)) * m_;
    B    += (8 * I2__ / (cx*cx*cy*cy)) * m_;

    // Right neighbour
    i_    = idx(hclampx(ix+1), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I1__ / (cx*cx)) * m_;
    B    += (8 * I2__ / (cx*cx*cx*cx)) * m_;
    B    += (8 * I2__ / (cx*cx*cy*cy)) * m_;

    // Below neighbour
    i_    = idx(ix, lclampy(iy-1), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I1__ / (cy*cy)) * m_;
    B    += (8 * I2__ / (cy*cy*cy*cy)) * m_;
    B    += (8 * I2__ / (cx*cx*cy*cy)) * m_;

    // Above neighbour
    i_    = idx(ix, hclampy(iy+1), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I1__ / (cy*cy)) * m_;
    B    += (8 * I2__ / (cy*cy*cy*cy)) * m_;
    B    += (8 * I2__ / (cx*cx*cy*cy)) * m_;


    /////////////////////////////////
    // Diagonal Nearest Neighbours //
    /////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(lclampx(ix-1), lclampy(iy-1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (4 * I2__ / (cx*cx*cy*cy)) * m_;

    // Top-left neighbour
    i_    = idx(lclampx(ix-1), hclampy(iy+1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (4 * I2__ / (cx*cx*cy*cy)) * m_;

    // Bottom-right neighbour
    i_    = idx(hclampx(ix+1), lclampy(iy-1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (4 * I2__ / (cx*cx*cy*cy)) * m_;

    // Top-right neighbour
    i_    = idx(hclampx(ix+1), hclampy(iy+1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (4 * I2__ / (cx*cx*cy*cy)) * m_;


    ///////////////////////////////////////
    // Next-Next-Next Nearest Neighbours //
    ///////////////////////////////////////

    // Two over to left
    i_    = idx(lclampx(ix-2), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cx*cx)) * m_;

    // Two over to right
    i_    = idx(hclampx(ix+2), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cx*cx)) * m_;

    // Two below
    i_    = idx(ix, lclampy(iy-2), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cy*cy*cy*cy)) * m_;

    // Two above
    i_    = idx(ix, hclampy(iy+2), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cy*cy*cy*cy)) * m_;


    float invMs = inv_Msat(Ms_, Ms_mul, I);

    Bx[I] += B.x*invMs;
    By[I] += B.y*invMs;
    Bz[I] += B.z*invMs;

}

