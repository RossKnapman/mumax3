#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange_fourth_order.go for more details.

extern "C" __global__ void
addexchangefourthorder(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ I1, float* __restrict__ I2,
            uint8_t* __restrict__ regions,
            float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int    I  = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3  B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float I1__;  // second-order exchange stiffness
    float I2__;  // fourth-order exchange stiffness


    //////////////////
    // Central Spin //
    //////////////////
    i_    = idx(ix, iy, iz);
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    += 2 * I1__ * (1/(cx*cx) + 1/(cy*cy) + 1/(cz*cz)) * m_;
    B    -= 6 * I2__ * (1/(cx*cx*cx*cx) + 1/(cy*cy*cy*cy) + 1/(cz*cz*cz*cz)) * m_;
    B    -= 8 * I2__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz) + 1/(cy*cy*cz*cz)) * m_;


    ///////////////////////////////
    // Direct Nearest Neighbours //
    ///////////////////////////////
    
    // Left neighbour
    i_    = idx(lclampx(ix-1), iy, iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I1__ / (cx*cx)) * m_;
    B    += 4 * I2__ * (1/(cx*cx*cx*cx) + 1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz)) * m_;

    // Right neighbour
    i_    = idx(hclampx(ix+1), iy, iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I1__ / (cx*cx)) * m_;
    B    += 4 * I2__ * (1/(cx*cx*cx*cx) + 1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz)) * m_;

    // Below neighbour
    i_    = idx(ix, lclampy(iy-1), iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I1__ / (cy*cy)) * m_;
    B    += 4 * I2__ * (1/(cy*cy*cy*cy) + 1/(cy*cy*cx*cx) + 1/(cy*cy*cz*cz)) * m_;

    // Above neighbour
    i_    = idx(ix, hclampy(iy+1), iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I1__ / (cy*cy)) * m_;
    B    += 4 * I2__ * (1/(cy*cy*cy*cy) + 1/(cy*cy*cx*cx) + 1/(cy*cy*cz*cz)) * m_;

    // Bottom neighbour
    i_    = idx(ix, iy, lclampz(iz-1));                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I1__ / (cz*cz)) * m_;
    B    += 4 * I2__ * (1/(cz*cz*cz*cz) + 1/(cz*cz*cx*cx) + 1/(cz*cz*cy*cy)) * m_;

    // Top neighbour
    i_    = idx(ix, iy, hclampz(iz+1));                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    I1__  = I1[symidx(r0, regions[i_])];
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I1__ / (cz*cz)) * m_;
    B    += 4 * I2__ * (1/(cz*cz*cz*cz) + 1/(cz*cz*cx*cx) + 1/(cz*cz*cy*cy)) * m_;


    //////////////////////////////////////////////
    // Diagonal Nearest Neighbours in z=0 Plane //
    //////////////////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(lclampx(ix-1), lclampy(iy-1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cy*cy)) * m_;

    // Top-left neighbour
    i_    = idx(lclampx(ix-1), hclampy(iy+1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cy*cy)) * m_;

    // Bottom-right neighbour
    i_    = idx(hclampx(ix+1), lclampy(iy-1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cy*cy)) * m_;

    // Top-right neighbour
    i_    = idx(hclampx(ix+1), hclampy(iy+1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cy*cy)) * m_;


    //////////////////////////////////////////////
    // Diagonal Nearest Neighbours in x=0 Plane //
    //////////////////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(ix, lclampy(iy-1), hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cy*cy*cz*cz)) * m_;

    // Top-left neighbour
    i_    = idx(ix, hclampy(iy+1), hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cy*cy*cz*cz)) * m_;

    // Bottom-right neighbour
    i_    = idx(ix, lclampy(iy-1), lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cy*cy*cz*cz)) * m_;

    // Top-right neighbour
    i_    = idx(ix, hclampy(iy+1), lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cy*cy*cz*cz)) * m_;


    //////////////////////////////////////////////
    // Diagonal Nearest Neighbours in y=0 Plane //
    //////////////////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(lclampx(ix-1), iy, hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cz*cz)) * m_;

    // Top-left neighbour
    i_    = idx(lclampx(ix-1), iy, lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cz*cz)) * m_;

    // Bottom-right neighbour
    i_    = idx(hclampx(ix+1), iy, hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cz*cz)) * m_;

    // Top-right neighbour
    i_    = idx(hclampx(ix+1), iy, lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (2 * I2__ / (cx*cx*cz*cz)) * m_;


    ///////////////////////////////////////
    // Next-Next-Next Nearest Neighbours //
    ///////////////////////////////////////

    // Two over to left
    i_    = idx(lclampx(ix-2), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I2__ / (cx*cx*cx*cx)) * m_;

    // Two over to right
    i_    = idx(hclampx(ix+2), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I2__ / (cx*cx*cx*cx)) * m_;

    // Two below
    i_    = idx(ix, lclampy(iy-2), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I2__ / (cy*cy*cy*cy)) * m_;

    // Two above
    i_    = idx(ix, hclampy(iy+2), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I2__ / (cy*cy*cy*cy)) * m_;

    // Two bottom
    i_    = idx(ix, iy, lclampz(iz-2));           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I2__ / (cz*cz*cz*cz)) * m_;

    // Two top
    i_    = idx(ix, iy, hclampz(iz+2));           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    I2__  = I2[symidx(r0, regions[i_])];
    B    -= (I2__ / (cz*cz*cz*cz)) * m_;


    float invMs = inv_Msat(Ms_, Ms_mul, I);

    Bx[I] += B.x*invMs;
    By[I] += B.y*invMs;
    Bz[I] += B.z*invMs;

}

