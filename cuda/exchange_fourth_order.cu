#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange_fourth_order.go for more details.

///////////////////////////////////////////////////
// Important note: Currently only works for Nz=1 //
///////////////////////////////////////////////////

extern "C" __global__ void
addexchangefourthorder(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ aSecondOrderLUT2d, float* __restrict__ aFourthOrderLUT2d,
            uint8_t* __restrict__ regions,
            float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int    I  = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3  B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float aSecondOrder__;  // second-order exchange stiffness
    float aFourthOrder__;  // fourth-order exchange stiffness


    //////////////////
    // Central Spin //
    //////////////////
    i_              = idx(ix, iy, iz);
    m_              = make_float3(mx[i_], my[i_], mz[i_]);                        // load m
    m_              = ( is0(m_)? m0: m_ );                                        // replace missing non-boundary neighbor
    aSecondOrder__  = aSecondOrderLUT2d[symidx(r0, regions[i_])];
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= 4 * aSecondOrder__ * (1/(cx*cx) + 1/(cy*cy)) * m_;
    B              -= 12 * aFourthOrder__ * (1/(cx*cx*cx*cx) + 1/(cy*cy*cy*cy)) * m_;
    B              -= 16 * aFourthOrder__ / (cx*cx*cy*cy) * m_;


    ///////////////////////////////
    // Direct Nearest Neighbours //
    ///////////////////////////////
    
    // Left neighbour
    i_              = idx(lclampx(ix-1), iy, iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aSecondOrder__  = aSecondOrderLUT2d[symidx(r0, regions[i_])];
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              += (2 * aSecondOrder__ / (cx*cx)) * m_;
    B              += (8 * aFourthOrder__ / (cx*cx*cx*cx)) * m_;
    B              += (8 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;

    // Right neighbour
    i_              = idx(hclampx(ix+1), iy, iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aSecondOrder__  = aSecondOrderLUT2d[symidx(r0, regions[i_])];
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              += (2 * aSecondOrder__ / (cx*cx)) * m_;
    B              += (8 * aFourthOrder__ / (cx*cx*cx*cx)) * m_;
    B              += (8 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;

    // Below neighbour
    i_              = idx(ix, lclampy(iy-1), iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aSecondOrder__  = aSecondOrderLUT2d[symidx(r0, regions[i_])];
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              += (2 * aSecondOrder__ / (cy*cy)) * m_;
    B              += (8 * aFourthOrder__ / (cy*cy*cy*cy)) * m_;
    B              += (8 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;

    // Above neighbour
    i_              = idx(ix, hclampy(iy+1), iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aSecondOrder__  = aSecondOrderLUT2d[symidx(r0, regions[i_])];
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              += (2 * aSecondOrder__ / (cy*cy)) * m_;
    B              += (8 * aFourthOrder__ / (cy*cy*cy*cy)) * m_;
    B              += (8 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;


    /////////////////////////////////
    // Diagonal Nearest Neighbours //
    /////////////////////////////////

    // Bottom-left neighbour
    i_              = idx(lclampx(ix-1), lclampy(iy-1), iz);       // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (4 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;

    // Top-left neighbour
    i_              = idx(lclampx(ix-1), hclampy(iy+1), iz);       // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (4 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;

    // Bottom-right neighbour
    i_              = idx(hclampx(ix+1), lclampy(iy-1), iz);       // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (4 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;

    // Top-right neighbour
    i_              = idx(hclampx(ix+1), hclampy(iy+1), iz);       // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (4 * aFourthOrder__ / (cx*cx*cy*cy)) * m_;


    ///////////////////////////////////////
    // Next-Next-Next Nearest Neighbours //
    ///////////////////////////////////////

    // Two over to left
    i_              = idx(lclampx(ix-2), iy, iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (2 * aFourthOrder__ / (cx*cx*cx*cx)) * m_;

    // Two over to right
    i_              = idx(hclampx(ix+2), iy, iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (2 * aFourthOrder__ / (cx*cx*cx*cx)) * m_;

    // Two below
    i_              = idx(ix, lclampy(iy-2), iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (2 * aFourthOrder__ / (cy*cy*cy*cy)) * m_;

    // Two above
    i_              = idx(ix, hclampy(iy+2), iz);                  // clamps or wraps index according to PBC
    m_              = make_float3(mx[i_], my[i_], mz[i_]);         // load m
    m_              = ( is0(m_)? m0: m_ );                         // replace missing non-boundary neighbor
    aFourthOrder__  = aFourthOrderLUT2d[symidx(r0, regions[i_])];
    B              -= (2 * aFourthOrder__ / (cy*cy*cy*cy)) * m_;


    float invMs = inv_Msat(Ms_, Ms_mul, I);

    Bx[I] += B.x*invMs;
    By[I] += B.y*invMs;
    Bz[I] += B.z*invMs;
}

